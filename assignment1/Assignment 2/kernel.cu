#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <time.h>
#include <math.h>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include "hip/hip_texture_types.h"
#include "texture_fetch_functions.hpp"


#define FAILURE 0
#define SUCCESS !FAILURE
#define _CRT_SECURE_NO_WARNINGS
#define HEADCHAR 24
#define USER_NAME "acp18ts"		//replace with your user name
#define THREADS 16
#define CUDA_CALL(x) {hipError_t cuda_error__ = (x); if (cuda_error__) printf("CUDA error: " #x " returned \"%s\"\n", hipGetErrorString(cuda_error__));}
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef struct {
	unsigned char r, g, b;
} Pixel;

typedef struct {
	int r, g, b;
} Pixel2;

typedef struct {
	unsigned int height, width, magic, max_val;
	Pixel2 *data;
} Image;

typedef struct {
	int start, stop;
} Timer;
typedef enum MODE { CPU, OPENMP, CUDA, ALL } MODE;

void print_help();
int process_command_line(int argc, char *argv[]);
static Image *import_ppm_hdr(char *imagein, char *binmode);
int powerof2(unsigned int c);
static Image *pad_array(Image *im, unsigned int c, Pixel2 *glob_av);
static Image *seq_mosaic_filter(Image *im, unsigned int c);
void output_ppm(char *filename, Image *im);
static Image *omp_mosaic_filter(Image *im, unsigned int c);
static Pixel2 *seq_print_av_col(Image *im);
static Pixel2 *omp_print_av_col(Image *im);
static Pixel2 cuda_print_av_col(Image *im);
static Image cuda_mosaic_filter2(Image *im, int mosaic_dim);


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

unsigned int c = 0;
MODE execution_mode = CPU;

char *mode;
char *imagein;
char *imageout;
char *binmode;

int main(int argc, char *argv[]) {

	// Checking command line inputs for user error
	if (process_command_line(argc, argv) == FAILURE)
		return 1;
	if (powerof2(c) != 1) {
		printf("%d is not a power of 2!", c);
		exit(2);
	}
	//TODO: read input image file (either binary or plain text PPM) 
	Image *im_in;
	im_in = (Image *)malloc(sizeof(Image));
	im_in = import_ppm_hdr(imagein, binmode);

	// checking C is not bigger than image dims
	if (c > im_in->height || c > im_in->width) {
		printf("%d is bigger than image dimensions of %d by %d", c, im_in->height, im_in->width);
		exit(3);
	}



	//TODO: execute the mosaic filter based on the mode
	if (strcmp(mode, "CPU") == 0) {
		execution_mode = CPU;
	}
	else if (strcmp(mode, "OPENMP") == 0) {
		execution_mode = OPENMP;
	}
	else if (strcmp(mode, "CUDA") == 0) {
		execution_mode = CUDA;
	}
	else if (strcmp(mode, "ALL") == 0) {
		execution_mode = ALL;

	}
	else {
		printf("execution mode %s is not recognised.", mode);
		exit(11);
	}
	switch (execution_mode) {
	case (CPU): {

		clock_t start, stop;
		start = clock();
		Pixel2 *glob_av;
		glob_av = (Pixel2*)malloc(sizeof(Pixel2));
		glob_av = seq_print_av_col(im_in);
		//Start the mosaic filter
		im_in = pad_array(im_in, c, glob_av);
		im_in = seq_mosaic_filter(im_in, c);
		//TODO: end timing here
		stop = clock();
		printf("CPU mode execution time took %.4fs\n", ((double)(stop - start) / CLOCKS_PER_SEC));

		break;
	}
	case (OPENMP): {
		//TODO: starting timing here
		clock_t start, stop;
		start = clock();
		//TODO: calculate the average colour value
		Pixel2 *glob_av;
		glob_av = (Pixel2*)malloc(sizeof(Pixel2));
		glob_av = omp_print_av_col(im_in);
		// Output the average colour value for the image
		im_in = pad_array(im_in, c, glob_av);
		im_in = omp_mosaic_filter(im_in, c);
		//TODO: end timing here
		stop = clock();
		printf("OPENMP mode execution time took %.4fs\n", ((double)(stop - start) / CLOCKS_PER_SEC));
		break;
	}
	case (CUDA): {
		clock_t start, stop;
		//TODO: starting timing here
		start = clock();
		//TODO: calculate the average colour value
		Pixel2 cuda_av = cuda_print_av_col(im_in);
		// Output the average colour value for the image
		Image cuda_im;
		cuda_im = cuda_mosaic_filter2(im_in, c); im_in = &cuda_im;

		//TODO: end timing here
		stop = clock();
		printf("CUDA mode execution time took %.4fs\n", ((double)(stop - start) / CLOCKS_PER_SEC));

		break;
	}
	case (ALL): {
		//TODO: starting timing here

		clock_t start, stop;
		start = clock();
		Pixel2 *glob_av;
		glob_av = (Pixel2*)malloc(sizeof(Pixel2));
		glob_av = seq_print_av_col(im_in);
		//Start the mosaic filter
		im_in = pad_array(im_in, c, glob_av);
		im_in = seq_mosaic_filter(im_in, c);
		//TODO: end timing here
		stop = clock();
		printf("CPU mode execution time took %.4fs\n", ((double)(stop - start) / CLOCKS_PER_SEC));


		//TODO: starting timing here
		//start = clock();
		//TODO: calculate the average colour value
		//glob_av = omp_print_av_col(im_in);
		// Output the average colour value for the image
		//im_in = pad_array(im_in, c, glob_av);
		//im_in = omp_mosaic_filter(im_in, c);
		//TODO: end timing here
		//stop = clock();
		//printf("OPENMP mode execution time took %.4fs\n", ((double)(stop - start) / CLOCKS_PER_SEC));


		//TODO: starting timing here
		start = clock();
		//TODO: calculate the average colour value
		Pixel2 cuda_av = cuda_print_av_col(im_in);
		// Output the average colour value for the image
		Image cuda_im;
		cuda_im = cuda_mosaic_filter2(im_in, c); im_in = &cuda_im;

		//TODO: end timing here
		stop = clock();
		printf("CUDA mode execution time took %.4fs\n", ((double)(stop - start) / CLOCKS_PER_SEC));




		break;
	}
	}

	//save the output image file (from last executed mode)
	output_ppm(imageout, im_in);
	return 0;
}

void print_help() {
	printf("mosaic_%s C M -i input_file -o output_file [options]\n", USER_NAME);

	printf("where:\n");
	printf("\tC              Is the mosaic cell size which should be any positive\n"
		"\t               power of 2 number \n");
	printf("\tM              Is the mode with a value of either CPU, OPENMP, CUDA or\n"
		"\t               ALL. The mode specifies which version of the simulation\n"
		"\t               code should execute. ALL should execute each mode in\n"
		"\t               turn.\n");
	printf("\t-i input_file  Specifies an input image file\n");
	printf("\t-o output_file Specifies an output image file which will be used\n"
		"\t               to write the mosaic image\n");
	printf("[options]:\n");
	printf("\t-f ppm_format  PPM image output format either PPM_BINARY (default) or \n"
		"\t               PPM_PLAIN_TEXT\n ");
}

int process_command_line(int argc, char *argv[]) {
	if (argc < 7) {
		fprintf(stderr, "Error: Missing program arguments. Correct usage is...\n");
		print_help();
		return FAILURE;
	}

	//first argument is always the executable name

	//read in the non optional command line arguments
	c = (unsigned int)atoi(argv[1]);

	//TODO: read in the mode
	int _argc = 2;
	mode = argv[_argc++];
	printf("C is %d \n", c);
	printf("mode is: %s\n", mode);
	//Validate mode

	imageout = 0; // SET DEFUALTS
	imagein = 0;
	binmode = (char*)"PPM_BINARY";
	while (_argc < argc)
	{
		if (strcmp("-o", argv[_argc]) == 0) //WHEN -0 IS FOUND IN ARGS SET NEXT ARG TO OUPUT
		{
			imageout = argv[++_argc];
			printf("imageout: %s\n", imageout);
		}
		else if (strcmp("-i", argv[_argc]) == 0) //WHEN -I IS FOUND IN ARGS SET NEXT ARG TO INPUT
		{
			imagein = argv[++_argc];
			printf("imagein: %s\n", imagein);
		}
		else if (strcmp("-f", argv[_argc]) == 0) //WHEN -F IS FOUND SET NEXT ARG TO BINARY MODE
		{
			binmode = argv[++_argc];
			printf("binary mode: %s\n", binmode);
		}
		else
		{
			printf("Unexpected arg: %s\n", argv[_argc]);
			return FAILURE;
		}
		++_argc;
	}
	if (imageout == 0 || imagein == 0)
		return FAILURE;
	return SUCCESS;
}

static Image *import_ppm_hdr(char *imagein, char *binmode) {


	Image *im; // define image struct called img
	FILE *file_point;
	char magic[64]; // define 64 long array for first line of header, 64 incase there is a long comment before first \n
	char temp[64];
	int width, height, max_val;
	int i = 0;

	if (strcmp(binmode, "PPM_BINARY") == 0) {
		file_point = fopen(imagein, "rb");
		if (file_point == NULL) {
			printf("Could not open file %s", imagein);
			exit(4);
		}
	}
	else if (strcmp(binmode, "PPM_PLAIN_TEXT") == 0) {
		file_point = fopen(imagein, "r");
		if (file_point == NULL) {
			printf("Could not open file %s", imagein);
			exit(4);
		}
	}
	else {
		printf("%s is not a valid PPM mode", binmode);
		exit(10);
	}


	fgets(magic, sizeof(magic), file_point);
	while (magic[0] != 'P' && i < 20) {
		fgets(magic, sizeof(magic), file_point);
		i++;
	}
	if (magic[0] != 'P') {
		printf("%s is not a PPM file or there are too many comments before header data", imagein);
		exit(5);
	}

	//check the image format
	if (magic[1] != '3' && magic[1] != '6') { // makes sure first line is magic num p6 for bin, p9 for ascii
		fprintf(stderr, "Image format must be 'P6' or 'P3\n");
		printf("magic num is %c %c", magic[0], magic[1]);
		exit(6);
	}

	im = (Image *)malloc(sizeof(Image));
	//skip any potential comments 
	fgets(temp, sizeof(temp), file_point);
	while (temp[0] == '#') {
		fgets(temp, sizeof(temp), file_point);
	}
	sscanf(temp, "%d", &width); // assign next found number to width

								//skip any potential comments 
	fgets(temp, sizeof(temp), file_point);
	while (temp[0] == '#') {
		fgets(temp, sizeof(temp), file_point);
	}
	sscanf(temp, "%d", &height); // assign next found number to height

								 //skip any potential comments 
	fgets(temp, sizeof(temp), file_point);
	while (temp[0] == '#') {
		fgets(temp, sizeof(temp), file_point);
	}
	sscanf(temp, "%d \n", &max_val); // assign next found number to max

								  //while (fgetc(file_point) != '\n');

	Pixel* data;
	data = (Pixel*)malloc(width * height * sizeof(Pixel)); //memory needed for image = number of elements * memory size of 1 pixel
	Pixel2* data_int;
	data_int = (Pixel2*)malloc(width * height * sizeof(Pixel2));

	if (!im) { // check if mem allocation fails
		printf("Unable to allocate memory for image data \n");
		exit(7);
	}

	//read pixel data from file
	if (strcmp(binmode, "PPM_BINARY") == 0) {
		if (fread(data, sizeof(Pixel), width*height, file_point) != width * height) { // reads into img data, the number of elements to be read is 3 (rgb) * number of x elements because each element in the array is a y row. Thus third arg is the size of a y row.
			fprintf(stderr, "Error loading image '%s'\n", imagein);
			exit(8);
		}
	}

	else if (strcmp(binmode, "PPM_PLAIN_TEXT") == 0) {
		for (i = 0; i < width * height; i++) {
			fscanf(file_point, "%c %c %c", &data[i].r, &data[i].g, &data[i].b);
		}
	}


	int numel = width * height;

	for (i = 0; i < numel; i++) {
		data_int[i].r = data[i].r;
		data_int[i].g = data[i].g;
		data_int[i].b = data[i].b;
	}

	im->magic = magic[1] - '0';
	im->height = height;
	im->width = width;
	im->max_val = max_val;
	im->data = data_int;

	fclose(file_point);
	return im;
}

int powerof2(unsigned int c)
{
	while (((c % 2) == 0) && c > 1)
		c /= 2;
	return (c == 1);
}

static Image *pad_array(Image *im, unsigned int c, Pixel2 *glob_av) {

	unsigned int rows_required, cols_required;
	rows_required = (unsigned int)ceil((double)im->height / c);
	cols_required = (unsigned int)ceil((double)im->width / c);
	unsigned int windows_width = cols_required * c;
	unsigned int windows_height = rows_required * c;
	int pad_sides = (int)ceil((float)(windows_width - im->width) / 2);
	int pad_height = (int)ceil((float)(windows_height - im->height) / 2);

	if (pad_height != 0 || pad_sides != 0) {
		Pixel2* padded;
		padded = (Pixel2*)malloc(windows_width * windows_height * sizeof(Pixel2));
		unsigned int i, j;
		for (i = 0; i < windows_width * windows_height; i++) {
			padded[i] = *glob_av;
		}
		int idx, idx_pad;
		for (i = 0; i < im->width; i++) {
			for (j = 0; j < im->height; j++) {
				idx = i * im->width + j;
				idx_pad = (i + pad_sides)*(im->width + pad_sides) + (j + pad_height);
				padded[idx_pad] = im->data[idx];


			}
		}
		im->data = padded;
		im->height = windows_height;
		im->width = 1337;
	}
	return(im);
}

static Image *seq_mosaic_filter(Image *im, unsigned int c) {
	// need to find indices of where mosiac windows will be applied so we can iterate
	unsigned int rows_required, cols_required;
	rows_required = im->height / c;
	cols_required = im->width / c;
	int *height_idxs, *width_idxs;
	unsigned int i = 0, j = 0, k = 0, l = 0;
	height_idxs = (int*)malloc(rows_required * sizeof(int));
	width_idxs = (int*)malloc(cols_required * sizeof(int));
	for (i = 0; i < rows_required; i++) {
		height_idxs[i] = c * i;
	}
	for (i = 0; i < cols_required; i++) {
		width_idxs[i] = c * i;

	}

	int *window_idxs;
	window_idxs = (int*)malloc(c * c * sizeof(int));

	int col_idx, row_idx, im_idx, relative_index, write_idx;
	double local_r = 0, local_g = 0, local_b = 0, c_dub = (double)c;
	Pixel2 *local_av;
	local_av = (Pixel2*)malloc(sizeof(Pixel2));
	local_av->r = 0; local_av->g = 0; local_av->b = 0;

	for (i = 0; i < cols_required; i++) {
		col_idx = i * c * im->width; //i*c to skip to next window index and then * width to account for flattened array
		for (j = 0; j < rows_required; j++) {

			row_idx = (j * c);
			im_idx = col_idx + row_idx;
			// iterating through idx of every top right corner of windows
			for (k = 0; k < c; k++) {
				for (l = 0; l < c; l++) {
					relative_index = (k*im->width) + l;
					window_idxs[k*c + l] = im_idx + relative_index;
					local_r = local_r + (double)im->data[im_idx + (k * im->width) + l].r;
					local_g = local_g + (double)im->data[im_idx + (k * im->width) + l].g;
					local_b = local_b + (double)im->data[im_idx + (k * im->width) + l].b;
				}
			}
			local_r = local_r / (c_dub*c_dub);
			local_g = local_g / (c_dub*c_dub);
			local_b = local_b / (c_dub*c_dub);

			local_av->r = (int)round(local_r);
			local_av->g = (int)round(local_g);
			local_av->b = (int)round(local_b);
			local_r = 0, local_g = 0, local_b = 0;
			for (k = 0; k < c*c; k++) { // iterate through the indexes we stored when we were calculating the window average and assign averge values into the image.
				write_idx = window_idxs[k];
				im->data[write_idx] = *local_av;
			}

		}
	}
	return(im);
}

void output_ppm(char *filename, Image *im) {
	FILE *file_point;
	//open file for output
	file_point = fopen(filename, "w");

	if (!file_point) {
		fprintf(stderr, "Unable to open file '%s'\n", filename);
		exit(12);
	}

	//write the header file
	//image format
	fprintf(file_point, "P6\n");


	//image size
	fprintf(file_point, "%d\n%d\n", im->width, im->height);

	// rgb component depth
	fprintf(file_point, "%d\n", im->max_val);

	Pixel *char_data;
	char_data = (Pixel*)malloc(im->width * im->height * sizeof(Pixel)); //memory needed for image = number of elements * memory size of 1 pixel
	for (unsigned int i = 0; i < im->height*im->width; i++) {
		char_data[i].r = (unsigned char)im->data[i].r;
		char_data[i].g = (unsigned char)im->data[i].g;
		char_data[i].b = (unsigned char)im->data[i].b;
	}

	// pixel data
	/*unsigned int i;
	for (i = 0; i < im->width * im->height; i++) {
	fprintf(file_point, "%c %c %c ", char_data[i].r, char_data[i].g, char_data[i].b);
	}*/

	fwrite(char_data, sizeof(Pixel), im->height*im->width, file_point);
	fclose(file_point);
}

static Image *omp_mosaic_filter(Image *im, unsigned int c) {
	// need to find indices of where mosiac windows will be applied so we can iterate
	int rows_required, cols_required;
	rows_required = im->height / c;
	cols_required = im->width / c;
	int *height_idxs, *width_idxs;
	height_idxs = (int*)malloc(rows_required * sizeof(int));
	width_idxs = (int*)malloc(cols_required * sizeof(int));
	int n;
	double c_2 = (double)c*c;


	for (n = 0; n < rows_required; n++) {
		height_idxs[n] = c * n;
	}
	for (n = 0; n < cols_required; n++) {
		width_idxs[n] = c * n;

	}

#pragma omp parallel 
	{
		int i = 0, j = 0;
		int *window_idxs;
		window_idxs = (int*)malloc(c * c * sizeof(int));

#pragma omp parallel for 
		for (i = 0; i < cols_required; i++) {
			for (j = 0; j < rows_required; j++) {
				Pixel2 local_av;
				double local_r = 0, local_g = 0, local_b = 0;
				unsigned int k = 0, l = 0;

				int col_idx, row_idx, im_idx, relative_index, write_idx;

				col_idx = i * c * im->width; //i*c to skip to next window index and then * width to account for flattened array

				local_av.r = 0; local_av.g = 0; local_av.b = 0;
				row_idx = (j * c);
				im_idx = col_idx + row_idx;

				// iterating through idx of every top right corner of windows
				for (k = 0; k < c; k++) {
					for (l = 0; l < c; l++) {
						relative_index = (k*im->width) + l;
						window_idxs[k*c + l] = im_idx + relative_index;
						local_r = local_r + (double)im->data[im_idx + relative_index].r;
						local_g = local_g + (double)im->data[im_idx + relative_index].g;
						local_b = local_b + (double)im->data[im_idx + relative_index].b;
					}
				}
				local_r = round(local_r / (c_2));
				local_g = round(local_g / (c_2));
				local_b = round(local_b / (c_2));

				local_av.r = (int)local_r;
				local_av.g = (int)local_g;
				local_av.b = (int)local_b;
				local_r = 0, local_g = 0, local_b = 0;

				for (k = 0; k < c*c; k++) { // iterate through the indexes we stored when we were calculating the window average and assign averge values into the image.
					write_idx = window_idxs[k];
					im->data[write_idx] = local_av;
				}

			}
		}
	}
	return(im);
}

static Pixel2 *seq_print_av_col(Image *im) {
	unsigned int i, j;

	//TODO: calculate the average colour value
	double rglob_av = 0, bglob_av = 0, gglob_av = 0, n = (double)im->height * im->width;
	// we access width as second index
	unsigned int idx;
	for (i = 0; i < im->height; i++) {
		for (j = 0; j < im->width; j++) {
			idx = (i*im->width) + j;
			rglob_av = rglob_av + (double)im->data[idx].r;
			gglob_av = gglob_av + (double)im->data[idx].g;
			bglob_av = bglob_av + (double)im->data[idx].b;
		}
	}
	rglob_av = round(rglob_av / n);
	gglob_av = round(gglob_av / n);
	bglob_av = round(bglob_av / n);
	// Output the average colour value for the image
	Pixel2 *glob_av;
	glob_av = (Pixel2*)malloc(sizeof(Pixel2));
	glob_av->r = (int)rglob_av;
	glob_av->g = (int)gglob_av;
	glob_av->b = (int)bglob_av;
	printf("CPU Average image colour red = %d, green = %d, blue = %d \n", glob_av->r, glob_av->g, glob_av->b);
	return glob_av;
}

static Pixel2 *omp_print_av_col(Image *im) {

	double rglob_av = 0, bglob_av = 0, gglob_av = 0, n = (double)im->height * im->width;

	int i, j, idx;

#pragma omp parallel for
	for (i = 0; i < (int)im->height; i++) {
		double tempr = 0, tempg = 0, tempb = 0;
		for (j = 0; j < (int)im->width; j++) {
			idx = (i*im->width) + j;
			if (idx == 0) {
				tempr = (double)im->data[idx].r;
				tempg = (double)im->data[idx].g;
				tempb = (double)im->data[idx].b;
			}
			else {
				tempr = tempr + (double)im->data[idx].r;
				tempg = tempg + (double)im->data[idx].g;
				tempb = tempb + (double)im->data[idx].b;
			}
		}
#pragma omp critical
		{
			rglob_av = rglob_av + tempr;
			gglob_av = gglob_av + tempg;
			bglob_av = bglob_av + tempb;

		}
	}
	rglob_av = rglob_av / n;
	gglob_av = gglob_av / n;
	bglob_av = bglob_av / n;

	Pixel2 *glob_av;
	glob_av = (Pixel2*)malloc(sizeof(Pixel2));
	glob_av->r = (int)round(rglob_av);
	glob_av->g = (int)round(gglob_av);
	glob_av->b = (int)round(bglob_av);
	printf("OMP Average image colour red = %d, green = %d, blue = %d \n", glob_av->r, glob_av->g, glob_av->b);
	return glob_av;
}

__global__ void cuda_av_kern(Image *im, Pixel2 *av, int *numel) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int tempr = 0, tempg = 0, tempb = 0;

	if (idx < *numel) {
		tempr = im->data[idx].r;
		tempg = im->data[idx].g;
		tempb = im->data[idx].b;

		int *r, *g, *b;
		r = &av->r;
		g = &av->g;
		b = &av->b;

		tempr = atomicAdd(r, tempr);
		tempg = atomicAdd(g, tempg);
		tempb = atomicAdd(b, tempb);
		//printf("the value being added is %d %d %d \n", tempr, tempg, tempb);
	}

}

__global__ void cuda_mosaic_kern(Image *im, int *numel) {

	int skip_line = im->width;
	int block_start = (blockIdx.x*blockDim.x) + (blockIdx.y*blockDim.y*skip_line);
	int thread_relative = threadIdx.x + (threadIdx.y*skip_line);
	int idx = block_start + thread_relative;
	//int block_size = blockDim.x*blockDim.y

	int truey = blockIdx.y*blockDim.y + threadIdx.y;
	int truex = blockIdx.x*blockDim.x + threadIdx.x;


	int tempr = 0, tempg = 0, tempb = 0;
	__shared__  int mosaic_r, mosaic_g, mosaic_b, tally;

	/*if (idx == 0) {
		mosaic_r = 0; mosaic_g = 0; mosaic_b = 0;
	}*/
	__syncthreads();

	if (truey < im->height && truex < im->width) {
		tempr = im->data[idx].r;
		tempg = im->data[idx].g;
		tempb = im->data[idx].b;

		int *r, *g, *b, *t;
		r = &mosaic_r;
		g = &mosaic_g;
		b = &mosaic_b;
		t = &tally;

		atomicAdd(r, tempr);
		atomicAdd(g, tempg);
		atomicAdd(b, tempb);
		atomicAdd(t, 1);
	}
	__syncthreads();
	if (truey < im->height && truex < im->width) {
		im->data[idx].r = (int)round((double)mosaic_r / (double)(tally));
		im->data[idx].g = (int)round((double)mosaic_g / (double)(tally));
		im->data[idx].b = (int)round((double)mosaic_b / (double)(tally));

	}
	
}

//__global__ void cuda_mosaic_kern3(Image *im, int *numel) {
//
//	int skip_line = im->width;
//	int block_start = (blockIdx.x*blockDim.x) + (blockIdx.y*blockDim.y*skip_line);
//	int thread_relative = threadIdx.x + (threadIdx.y*skip_line);
//	int idx = block_start + thread_relative;
//	int tid = threadIdx.x + threadIdx.y*blockDim.x;
//
//	int truey = blockIdx.y*blockDim.y + threadIdx.y;
//	int truex = blockIdx.x*blockDim.x + threadIdx.x;
//
//	int tempr = 0, tempg = 0, tempb = 0;
//	extern __shared__  int s_mem[];
//	int size_share = sizeof(s_mem);
//	int*   mosaic_r = (int*)&s_mem[64];
//	__syncthreads();
//
//	if (truey < im->height && truex < im->width) {
//		mosaic_r[tid] = im->data[idx].r;
//		mosaic_g[tid] = im->data[idx].g;
//		mosaic_b[tid] = im->data[idx].b;
//		
//	}
//	__syncthreads();
//	// do reduction in shared mem
//	/*if (truey < im->height && truex < im->width) {
//		for (unsigned int s = 1; s < blockDim.x; s *= 2) {
//			if (tid % (2 * s) == 0) {
//				mosaic_r[tid] += mosaic_r[tid + s];
//				mosaic_g[tid] += mosaic_g[tid + s];
//				mosaic_b[tid] += mosaic_b[tid + s];
//			}
//		}
//	}*/
//	__syncthreads();
//	
//	if (truey < im->height && truex < im->width) {
//		im->data[idx].r = (int)round((double)mosaic_r[0] / (double)(1024));
//		im->data[idx].g = (int)round((double)mosaic_g[0] / (double)(1024));
//		im->data[idx].b = (int)round((double)mosaic_b[0] / (double)(1024));
//	}
//
//}

__global__ void cuda_mosaic_kern4(Image *im, int *numel) {

	int skip_line = im->width;
	int block_start = (blockIdx.x*blockDim.x) + (blockIdx.y*blockDim.y*skip_line);
	int thread_relative = threadIdx.x + (threadIdx.y*skip_line);
	int idx = block_start + thread_relative;

	int truey = blockIdx.y*blockDim.y + threadIdx.y;
	int truex = blockIdx.x*blockDim.x + threadIdx.x;

	int tempr = 0, tempg = 0, tempb = 0;
	extern __shared__  Pixel2 mosaic_av;
	__syncthreads();

	if (truey < im->height && truex < im->width) {
		tempr = im->data[idx].r;
		tempg = im->data[idx].g;
		tempb = im->data[idx].b;

		int *r, *g, *b;
		r = &mosaic_av.r;
		g = &mosaic_av.g;
		b = &mosaic_av.b;

		atomicAdd(r, tempr);
		atomicAdd(g, tempg);
		atomicAdd(b, tempb);
	}
	__syncthreads();
	if (truey < im->height && truex < im->width) {
		im->data[idx].r = (int)round((double)mosaic_av.r / (double)(1024));
		im->data[idx].g = (int)round((double)mosaic_av.g / (double)(1024));
		im->data[idx].b = (int)round((double)mosaic_av.b / (double)(1024));
	}
}

__global__ void cuda_simple_mosaic_kern(Image *im, int *numel) {
	// secify variables we need for calculating indexes
	int skipline = im->width;
	int mosaic_start = (threadIdx.x*blockDim.x) + (threadIdx.y*blockDim.y*skipline);
	int relative_index = 0, absolute_index = 0;
	int i, j;

	int tile_av_r = 0, tile_av_g = 0, tile_av_b = 0;

	// iterate through mosaic accumulating values
	for (i = 0; i >= blockDim.x; i++) {
		for (j = 0; j >= blockDim.y; j++) {
			relative_index = i + j * skipline;
			absolute_index = mosaic_start + relative_index;
			if (absolute_index > *numel) {
				tile_av_r += im->data[absolute_index].r;
				tile_av_g += im->data[absolute_index].g;
				tile_av_b += im->data[absolute_index].b;
			}
		}
	}

	// calculate mean of all R, G and B values
	tile_av_r = (int)round((double)tile_av_r / (double)(blockDim.x*blockDim.y));
	tile_av_g = (int)round((double)tile_av_g / (double)(blockDim.x*blockDim.y));
	tile_av_b = (int)round((double)tile_av_b / (double)(blockDim.x*blockDim.y));

	// reiterate and assign averages to pixels.
	for (i = 0; i >= blockDim.x; i++) {
		for (j = 0; j >= blockDim.y; j++) {
			relative_index = i + j * skipline;
			absolute_index = mosaic_start + relative_index;
			if (absolute_index > *numel) {
				im->data[absolute_index].r = tile_av_r;
				im->data[absolute_index].g = tile_av_g;
				im->data[absolute_index].b = tile_av_b;
			}
		}
	}
}

__global__ void cuda_inter_mosaic_kern(Image *im, Pixel2 *data, Pixel2 *tile_averages) {
	// secify variables we need for calculating indexes
	int skipline = im->width;
	int mosaic_start = (blockIdx.x*blockDim.x) + (blockIdx.y*blockDim.y*skipline);
	int thread_relative = threadIdx.x + (threadIdx.y*skipline);
	int idx = mosaic_start + thread_relative;
	int tile_avs_idx = blockIdx.x + (blockIdx.y*gridDim.x);

	int truey = blockIdx.y*blockDim.y + threadIdx.y;
	int truex = blockIdx.x*blockDim.x + threadIdx.x;

	int tempr = 0, tempg = 0, tempb = 0;

	if (truey < im->height && truex < im->width) {
		tempr = data[idx].r;
		tempg = data[idx].g;
		tempb = data[idx].b;

		int *r, *g, *b;
		r = &tile_averages[tile_avs_idx].r;
		g = &tile_averages[tile_avs_idx].g;
		b = &tile_averages[tile_avs_idx].b;

		atomicAdd(r, tempr);
		atomicAdd(g, tempg);
		atomicAdd(b, tempb);



	}
	__syncthreads();
	if (truey < im->height && truex < im->width) {
		data[idx] = tile_averages[tile_avs_idx];
	}

}

static Pixel2 cuda_print_av_col(Image *im) {
	Image h_image;
	Image hd_image;
	Image *d_image;

	h_image = *im;
	hd_image = h_image;

	Pixel2 av;
	av.r = 0; av.g = 0; av.b = 0;
	Pixel2 *d_av;

	int numel = h_image.width*h_image.height;
	int N = (int)round((double)numel / (double)32 );
	int *d_numel;

	//Moving image data to device
	gpuErrchk(hipMalloc(&hd_image.data, h_image.width*h_image.height * sizeof(Pixel2)));
	gpuErrchk(hipMemcpy(hd_image.data, h_image.data, h_image.width*h_image.height * sizeof(Pixel2), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&d_image, sizeof(Image)));
	gpuErrchk(hipMemcpy(d_image, &hd_image, sizeof(Image), hipMemcpyHostToDevice));

	//Moving Pixel for average to device
	gpuErrchk(hipMalloc(&d_av, sizeof(Pixel2)));
	gpuErrchk(hipMemcpy(d_av, &av, sizeof(Pixel2), hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc(&d_numel, sizeof(int)));
	gpuErrchk(hipMemcpy(d_numel, &numel, sizeof(int), hipMemcpyHostToDevice));

	dim3 threadsPerBlock(32, 1, 1); // static blocks of 32 for now

	dim3 blocksPerGrid(N, 1, 1); // uses minimum amount of 32 blocks

	cuda_av_kern << <blocksPerGrid, threadsPerBlock >> > (d_image, d_av, d_numel); // runs kernel
	hipError_t kern_error = hipGetLastError();
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(&av, d_av, sizeof(Pixel2), hipMemcpyDeviceToHost));

	int avr, avg, avb;
	avr = (int)round((double)av.r / (double)numel);
	avg = (int)round((double)av.g / (double)numel);
	avb = (int)round((double)av.b / (double)numel);

	printf("CUDA Average image colour red = %d, green = %d, blue = %d \n", avr, avg, avb);

	gpuErrchk(hipFree(hd_image.data));
	hd_image.data = 0;
	gpuErrchk(hipFree(d_image));
	d_image = 0;

	return av;
}

static Image cuda_mosaic_filter2(Image *im, int mosaic_dim) {
	Image h_image;
	Image hd_image;
	Image *d_image;

	h_image = *im;
	hd_image = h_image;


	int numel = h_image.width*h_image.height;
	int *d_numel;

	

	//Decide how to process based on dimensions
	if (h_image.height * h_image.width <= 1024 && mosaic_dim * mosaic_dim <= 1024) {
		//Moving image data to device
		gpuErrchk(hipMalloc((void **)&hd_image.data, h_image.width*h_image.height * sizeof(Pixel2)));
		gpuErrchk(hipMemcpy(hd_image.data, h_image.data, h_image.width*h_image.height * sizeof(Pixel2), hipMemcpyHostToDevice));
		gpuErrchk(hipMalloc(&d_image, sizeof(Image)));
		gpuErrchk(hipMemcpy(d_image, &hd_image, sizeof(Image), hipMemcpyHostToDevice));

		//moving numel int to device
		gpuErrchk(hipMalloc(&d_numel, sizeof(int)));
		gpuErrchk(hipMemcpy(d_numel, &numel, sizeof(int), hipMemcpyHostToDevice));



		//do 1 thread per mosaic tile
		int numy = (int)ceil((double)im->height / (double)c), numx = (int)ceil((double)im->width / (double)c);
		dim3 threadsPerBlock(numy, numx, 1);
		dim3 blocksPerGrid(1, 1, 1);
		// run kernel
		cuda_simple_mosaic_kern << < blocksPerGrid, threadsPerBlock >> > (d_image, d_numel);
		hipError_t kern_error = hipGetLastError();
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		//Bring back modified data from device to host
		gpuErrchk(hipMemcpy(&h_image, d_image, sizeof(Image), hipMemcpyDeviceToHost));
		Pixel2* out_image;
		out_image = (Pixel2*)malloc(h_image.height*h_image.width * sizeof(Pixel2));
		gpuErrchk(hipMemcpy(out_image, d_image->data, h_image.width*h_image.height * sizeof(Pixel2), hipMemcpyDeviceToHost));

		// free memory
		gpuErrchk(hipFree(hd_image.data));
		hd_image.data = 0;
		gpuErrchk(hipFree(d_image));
		d_image = 0;
	}

	else if (h_image.height * h_image.width >= 1024 && mosaic_dim * mosaic_dim <= 1024) {
		//1 block per mosaic tile
		Pixel2 *d_data;
		//calculating how many blocks we need and size of blocks for 1 block = 1 mosaic
		dim3 threadsPerBlock(mosaic_dim, mosaic_dim, 1);
		//int N = (int)round((double)numel / (double)(mosaic_dim*mosaic_dim));
		int numy = (int)ceil((double)im->height / (double)c), numx = (int)ceil((double)im->width / (double)c);
		dim3 blocksPerGrid(numx, numy, 1);

		Pixel2 *mosaic_vals, *d_mosaic_vals;
		mosaic_vals = (Pixel2*)malloc(numy*numx*sizeof(Pixel2));
		gpuErrchk(hipMalloc((void **)&d_mosaic_vals, numy*numx * sizeof(Pixel2)));
		gpuErrchk(hipMemset(d_mosaic_vals, 0, numy*numx * sizeof(Pixel2)));

		gpuErrchk(hipMalloc((void **)&d_data, h_image.width*h_image.height * sizeof(Pixel2)));
		gpuErrchk(hipMemcpy(d_data, im->data, h_image.width*h_image.height * sizeof(Pixel2), hipMemcpyHostToDevice));

		gpuErrchk(hipMalloc(&d_image, sizeof(Image)));
		gpuErrchk(hipMemcpy(d_image, &h_image, sizeof(Image), hipMemcpyHostToDevice));
		gpuErrchk(hipMalloc((void **)&d_mosaic_vals, numy*numx * sizeof(Pixel2)));

		// run kernel and check execution
		cuda_inter_mosaic_kern << <blocksPerGrid, threadsPerBlock >> > (d_image, d_data, d_mosaic_vals); // runs kernel
		hipError_t kern_error = hipGetLastError();
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());

		Pixel2 *h_data;
		h_data = (Pixel2*)malloc(h_image.width*h_image.height * sizeof(Pixel2));

		gpuErrchk(hipMemcpy(h_data, d_data, h_image.width*h_image.height * sizeof(Pixel2), hipMemcpyDeviceToHost));
		h_image.data = h_data;

		gpuErrchk(hipFree(d_data));
		gpuErrchk(hipFree(d_image));
		

	}



	return h_image;
}





